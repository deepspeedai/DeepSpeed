#include "hip/hip_runtime.h"
#include "cublas_wrappers.h"

int cublas_gemm_ex(gpu_lib::blasHandle_t handle,
                   gpu_lib::blasOperation_t transa,
                   gpu_lib::blasOperation_t transb,
                   int m,
                   int n,
                   int k,
                   const float* alpha,
                   const float* beta,
                   const float* A,
                   const float* B,
                   float* C,
                   gpu_lib::blasGemmAlgo_t algo)
{
    gpu_lib::blasStatus_t status = gpu_lib::blasGemmEx(handle,
                                                       transa,
                                                       transb,
                                                       m,
                                                       n,
                                                       k,
                                                       (const void*)alpha,
                                                       (const void*)A,
                                                       gpu_lib::DT_R_32F,
                                                       (transa == gpu_lib::BLAS_OP_N) ? m : k,
                                                       (const void*)B,
                                                       gpu_lib::DT_R_32F,
                                                       (transb == gpu_lib::BLAS_OP_N) ? k : n,
                                                       (const void*)beta,
                                                       C,
                                                       gpu_lib::DT_R_32F,
                                                       m,
                                                       gpu_lib::DT_R_32F,
                                                       algo);

    if (status != gpu_lib::BLAS_STATUS_SUCCESS) {
        fprintf(stderr,
                "!!!! kernel execution error. (m: %d, n: %d, k: %d, error: %d) \n",
                m,
                n,
                k,
                (int)status);
        return EXIT_FAILURE;
    }
    return 0;
}

int cublas_gemm_ex(gpu_lib::blasHandle_t handle,
                   gpu_lib::blasOperation_t transa,
                   gpu_lib::blasOperation_t transb,
                   int m,
                   int n,
                   int k,
                   const float* alpha,
                   const float* beta,
                   const __half* A,
                   const __half* B,
                   __half* C,
                   gpu_lib::blasGemmAlgo_t algo)
{
    gpu_lib::blasStatus_t status = hipblasGemmEx(handle,
                                                transa,
                                                transb,
                                                m,
                                                n,
                                                k,
                                                (const void*)alpha,
                                                (const void*)A,
                                                gpu_lib::DT_R_16F,
                                                (transa == gpu_lib::BLAS_OP_N) ? m : k,
                                                (const void*)B,
                                                gpu_lib::DT_R_16F,
                                                (transb == gpu_lib::BLAS_OP_N) ? k : n,
                                                (const void*)beta,
                                                (void*)C,
                                                gpu_lib::DT_R_16F,
                                                m,
                                                gpu_lib::DT_R_32F,
                                                algo);
    if (status != gpu_lib::BLAS_STATUS_SUCCESS) {
        fprintf(stderr,
                "!!!! kernel execution error. (m: %d, n: %d, k: %d, error: %d) \n",
                m,
                n,
                k,
                (int)status);
        return EXIT_FAILURE;
    }
    return 0;
}

#ifdef __HIP_PLATFORM_HCC__
int cublas_strided_batched_gemm(rocblas_handle handle,
                                int m,
                                int n,
                                int k,
                                const float* alpha,
                                const float* beta,
                                const float* A,
                                const float* B,
                                float* C,
                                rocblas_operation op_A,
                                rocblas_operation op_B,
                                int stride_A,
                                int stride_B,
                                int stride_C,
                                int batch,
                                rocblas_gemm_algo algo)
#else
int cublas_strided_batched_gemm(hipblasHandle_t handle,
                                int m,
                                int n,
                                int k,
                                const float* alpha,
                                const float* beta,
                                const float* A,
                                const float* B,
                                float* C,
                                hipblasOperation_t op_A,
                                hipblasOperation_t op_B,
                                int stride_A,
                                int stride_B,
                                int stride_C,
                                int batch,
                                hipblasGemmAlgo_t algo)
#endif
{
#ifdef __HIP_PLATFORM_HCC__
    rocblas_status status =
        rocblas_gemm_strided_batched_ex(handle,
                                        op_A,
                                        op_B,
                                        m,
                                        n,
                                        k,
                                        alpha,
                                        A,
                                        rocblas_datatype_f32_r,
                                        (op_A == rocblas_operation_none) ? m : k,
                                        stride_A,
                                        B,
                                        rocblas_datatype_f32_r,
                                        (op_B == rocblas_operation_none) ? k : n,
                                        stride_B,
                                        beta,
                                        C,
                                        rocblas_datatype_f32_r,
                                        m,
                                        stride_C,
                                        C,
                                        rocblas_datatype_f32_r,
                                        m,
                                        stride_C,
                                        batch,
                                        rocblas_datatype_f32_r,
                                        algo,
                                        0,
                                        0);
#else
    hipblasStatus_t status = hipblasGemmStridedBatchedEx(handle,
                                                       op_A,
                                                       op_B,
                                                       m,
                                                       n,
                                                       k,
                                                       alpha,
                                                       A,
                                                       HIP_R_32F,
                                                       (op_A == HIPBLAS_OP_N) ? m : k,
                                                       stride_A,
                                                       B,
                                                       HIP_R_32F,
                                                       (op_B == HIPBLAS_OP_N) ? k : n,
                                                       stride_B,
                                                       beta,
                                                       C,
                                                       HIP_R_32F,
                                                       m,
                                                       stride_C,
                                                       batch,
                                                       HIP_R_32F,
                                                       algo);
#endif

#ifdef __HIP_PLATFORM_HCC__
    if (status != rocblas_status_success) {
#else
    if (status != HIPBLAS_STATUS_SUCCESS) {
#endif
        fprintf(stderr,
                "!!!! kernel execution error. (batch: %d, m: %d, n: %d, k: %d, error: %d) \n",
                batch,
                m,
                n,
                k,
                (int)status);
        return EXIT_FAILURE;
    }
    return 0;
}

#ifdef __HIP_PLATFORM_HCC__
int cublas_strided_batched_gemm(rocblas_handle handle,
                                int m,
                                int n,
                                int k,
                                const float* alpha,
                                const float* beta,
                                const __half* A,
                                const __half* B,
                                __half* C,
                                rocblas_operation op_A,
                                rocblas_operation op_B,
                                int stride_A,
                                int stride_B,
                                int stride_C,
                                int batch,
                                rocblas_gemm_algo algo)
#else
int cublas_strided_batched_gemm(hipblasHandle_t handle,
                                int m,
                                int n,
                                int k,
                                const float* alpha,
                                const float* beta,
                                const __half* A,
                                const __half* B,
                                __half* C,
                                hipblasOperation_t op_A,
                                hipblasOperation_t op_B,
                                int stride_A,
                                int stride_B,
                                int stride_C,
                                int batch,
                                hipblasGemmAlgo_t algo)
#endif
{
#ifdef __HIP_PLATFORM_HCC__
    rocblas_status status =
        rocblas_gemm_strided_batched_ex(handle,
                                        op_A,
                                        op_B,
                                        m,
                                        n,
                                        k,
                                        alpha,
                                        A,
                                        rocblas_datatype_f16_r,
                                        (op_A == rocblas_operation_none) ? m : k,
                                        stride_A,
                                        B,
                                        rocblas_datatype_f16_r,
                                        (op_B == rocblas_operation_none) ? k : n,
                                        stride_B,
                                        beta,
                                        C,
                                        rocblas_datatype_f16_r,
                                        m,
                                        stride_C,
                                        C,
                                        rocblas_datatype_f16_r,
                                        m,
                                        stride_C,
                                        batch,
                                        rocblas_datatype_f32_r,
                                        algo,
                                        0,
                                        0);
#else
    hipblasStatus_t status = hipblasGemmStridedBatchedEx(handle,
                                                       op_A,
                                                       op_B,
                                                       m,
                                                       n,
                                                       k,
                                                       alpha,
                                                       A,
                                                       HIP_R_16F,
                                                       (op_A == HIPBLAS_OP_N) ? m : k,
                                                       stride_A,
                                                       B,
                                                       HIP_R_16F,
                                                       (op_B == HIPBLAS_OP_N) ? k : n,
                                                       stride_B,
                                                       beta,
                                                       C,
                                                       HIP_R_16F,
                                                       m,
                                                       stride_C,
                                                       batch,
                                                       HIP_R_32F,
                                                       algo);
#endif

#ifdef __HIP_PLATFORM_HCC__
    if (status != rocblas_status_success) {
#else
    if (status != HIPBLAS_STATUS_SUCCESS) {
#endif
        fprintf(stderr,
                "!!!! kernel execution error. (m: %d, n: %d, k: %d, error: %d) \n",
                m,
                n,
                k,
                (int)status);
        return EXIT_FAILURE;
    }

    return 0;
}
