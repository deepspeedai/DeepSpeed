#include "hip/hip_runtime.h"
/*
Copyright 2022 The Microsoft DeepSpeed Team
*/

#include "conversion_utils.h"
#include "inference_cuda_layers.h"
#include "memory_access_utils.h"

namespace cg = cooperative_groups;
#define MAX_CAP 4
#define MAX_SEQ 2048

inline __device__ float gelu(const float x)
{
    const float sqrt_param = 0.79788456080286535587989211986876f;
    const float mul_param = 0.044715;
    return x * 0.5f * (1.0f + tanhf(sqrt_param * (x + mul_param * x * x * x)));
}

template <typename T>
__global__ void fused_bias_gelu(T* input, const T* bias, int total_count, int intermediate_size)
{
    // Input restriction: intermediate_size % vals_per_access == 0
    constexpr int granularity = 16;
    constexpr int values_per_access = granularity / sizeof(T);
    const int offset = (blockIdx.x * blockDim.x + threadIdx.x) * values_per_access;

    if (offset < total_count) {
        T data[values_per_access];
        T data_bias[values_per_access];
        mem_access::load_global<granularity>(data, input + offset);
        mem_access::load_global<granularity>(data_bias, bias + (offset % intermediate_size));

#pragma unroll
        for (int i = 0; i < values_per_access; i++) {
            float data_f = conversion::to<float>(data[i]);
            float bias_f = conversion::to<float>(data_bias[i]);
            data[i] = conversion::to<T>(gelu(data_f + bias_f));
        }

        mem_access::store_global<granularity>(input + offset, data);
    }
}

template <typename T>
void launch_bias_gelu(T* input,
                      const T* bias,
                      int intermediate_size,
                      int batch_size,
                      hipStream_t stream)
{
    constexpr int threads = 1024;
    constexpr int granularity = 16;

    const int total_count = batch_size * intermediate_size;
    const int elems_per_block = threads * (granularity / sizeof(T));
    dim3 block_dims(threads);
    dim3 grid_dims((total_count + elems_per_block - 1) / elems_per_block);

    fused_bias_gelu<<<grid_dims, block_dims, 0, stream>>>(
        input, bias, total_count, intermediate_size);
}

template void launch_bias_gelu<float>(float*, const float*, int, int, hipStream_t);
template void launch_bias_gelu<__half>(__half*, const __half*, int, int, hipStream_t);

// Not called directly from DeepSpeed, but used in ds_qkv_gemm_int8, ds_linear_layer, etc.
__global__ void fused_bias_add(float* input, const float* bias, int total_count, int hidden_size)
{
    constexpr int granularity = 16;
    constexpr int vals_per_access = granularity / sizeof(float);
    const int offset = (blockIdx.x * blockDim.x + threadIdx.x) * vals_per_access;

    if (offset < total_count) {
        float data[vals_per_access];
        float bias_data[vals_per_access];
        mem_access::load_global<granularity>(data, input + offset);
        mem_access::load_global<granularity>(bias_data, bias + (offset % hidden_size));

#pragma unroll
        for (int i = 0; i < vals_per_access; i++) { data[i] += bias_data[i]; }

        mem_access::store_global<granularity>(input + offset, data);
    }
}

__global__ void fused_bias_add(__half* input, const __half* bias, int total_count, int hidden_size)
{
#ifdef HALF_PRECISION_AVAILABLE
    constexpr int granularity = 16;
    constexpr int vals_per_access = granularity / sizeof(__half);
    const int offset = (blockIdx.x * blockDim.x + threadIdx.x) * vals_per_access;

    if (offset < total_count) {
        __half2 data[vals_per_access / 2];
        __half2 bias_data[vals_per_access / 2];
        mem_access::load_global<granularity>(data, input + offset);
        mem_access::load_global<granularity>(bias_data, bias + (offset % hidden_size));

#pragma unroll
        for (int i = 0; i < vals_per_access / 2; i++) {
            float2 data_f = __half22float2(data[i]);
            float2 bias_f = __half22float2(bias_data[i]);
            data[i] = __floats2half2_rn(data_f.x + bias_f.x, data_f.y + bias_f.y);
        }

        mem_access::store_global<granularity>(input + offset, data);
    }
#endif
}

template <typename T>
void launch_bias_add(T* input, const T* bias, int hidden_size, int batch_size, hipStream_t stream)
{
    constexpr int threads = 1024;
    constexpr int granularity = 16;

    const int total_count = batch_size * hidden_size;
    const int elems_per_block = threads * (granularity / sizeof(T));
    dim3 block_dims(threads);
    dim3 grid_dims((total_count + elems_per_block - 1) / elems_per_block);

    fused_bias_add<<<grid_dims, block_dims, 0, stream>>>(input, bias, total_count, hidden_size);
}

template void launch_bias_add<float>(float*, const float*, int, int, hipStream_t);
template void launch_bias_add<__half>(__half*, const __half*, int, int, hipStream_t);

__global__ void fused_bias_residual(float* input,
                                    float* output,
                                    float* attn,
                                    float* bias,
                                    float* attnbias,
                                    int total_count,
                                    int intermediate_size,
                                    float mp_scale,
                                    bool preln)
{
    float4* input_cast = reinterpret_cast<float4*>(input);
    float4* output_cast = reinterpret_cast<float4*>(output);
    float4* attn_cast = reinterpret_cast<float4*>(attn);
    float4* bias_cast = reinterpret_cast<float4*>(bias);
    float4* attnbias_cast = reinterpret_cast<float4*>(attnbias);
    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    if (offset < total_count) {
        float4 data = input_cast[offset];
        float4 out = output_cast[offset];
        float4 res_vec = attn_cast[offset];
        float4 bias_data = bias_cast[offset % intermediate_size];
        float4 attn_bias = attnbias_cast[offset % intermediate_size];
        if (preln) {
            data.x = (data.x + res_vec.x + bias_data.x + attn_bias.x) * mp_scale + (out.x);
            data.y = (data.y + res_vec.y + bias_data.y + attn_bias.y) * mp_scale + (out.y);
            data.z = (data.z + res_vec.z + bias_data.z + attn_bias.z) * mp_scale + (out.z);
            data.w = (data.w + res_vec.w + bias_data.w + attn_bias.w) * mp_scale + (out.w);
        } else {
            data.x = data.x + out.x + bias_data.x;
            data.y = data.y + out.y + bias_data.y;
            data.z = data.z + out.z + bias_data.z;
            data.w = data.w + out.w + bias_data.w;
        }
        input_cast[offset] = data;
    }
}

__global__ void fused_bias_residual(__half* input,
                                    __half* output,
                                    __half* attn,
                                    __half* bias,
                                    __half* attn_bias,
                                    int total_count,
                                    int intermediate_size,
                                    float mp_scale,
                                    bool preln)
{
#ifdef HALF_PRECISION_AVAILABLE

    float2* input_cast = reinterpret_cast<float2*>(input);
    float2* output_cast = reinterpret_cast<float2*>(output);
    float2* attn_cast = reinterpret_cast<float2*>(attn);

    float2* bias_cast = reinterpret_cast<float2*>(bias);
    float2* attnbias_cast = reinterpret_cast<float2*>(attn_bias);

    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    if (offset < total_count) {
        float2 vals_vec = input_cast[offset];
        float2 out_vec = output_cast[offset];
        float2 res_vec = attn_cast[offset];

        float2 bias_vec = bias_cast[offset % intermediate_size];
        float2 attn_bias_vec = attnbias_cast[offset % intermediate_size];

        __half2* vals_half = reinterpret_cast<__half2*>(&vals_vec);
        __half2* out_half = reinterpret_cast<__half2*>(&out_vec);
        __half2* res_half = reinterpret_cast<__half2*>(&res_vec);
        __half2* bias_half = reinterpret_cast<__half2*>(&bias_vec);
        __half2* attnbias_half = reinterpret_cast<__half2*>(&attn_bias_vec);

        float2 low_data = __half22float2(vals_half[0]);
        float2 high_data = __half22float2(vals_half[1]);

        float2 low_out = __half22float2(out_half[0]);
        float2 high_out = __half22float2(out_half[1]);

        float2 low_res = __half22float2(res_half[0]);
        float2 high_res = __half22float2(res_half[1]);

        float2 low_bias = __half22float2(bias_half[0]);
        float2 high_bias = __half22float2(bias_half[1]);

        float2 attn_low_bias = __half22float2(attnbias_half[0]);
        float2 attn_high_bias = __half22float2(attnbias_half[1]);

        if (preln) {
            low_data.x =
                (low_data.x + low_res.x + (low_bias.x + attn_low_bias.x)) * mp_scale + low_out.x;
            low_data.y =
                (low_data.y + low_res.y + (low_bias.y + attn_low_bias.y)) * mp_scale + low_out.y;
            high_data.x = (high_data.x + high_res.x + (high_bias.x + attn_high_bias.x)) * mp_scale +
                          high_out.x;
            high_data.y = (high_data.y + high_res.y + (high_bias.y + attn_high_bias.y)) * mp_scale +
                          high_out.y;
        } else {
            low_data.x = (low_data.x + low_out.x + low_bias.x);
            low_data.y = (low_data.y + low_out.y + low_bias.y);
            high_data.x = (high_data.x + high_out.x + high_bias.x);
            high_data.y = (high_data.y + high_out.y + high_bias.y);
        }
        vals_half[0] = __float22half2_rn(low_data);
        vals_half[1] = __float22half2_rn(high_data);

        input_cast[offset] = vals_vec;
    }
#endif
}

template <typename T>
void launch_bias_residual(T* input,
                          T* output,
                          T* attn,
                          T* bias,
                          T* attn_bias,
                          int batch,
                          int hidden_dim,
                          int mp_size,
                          bool preln,
                          hipStream_t stream)
{
    int total_count = batch * hidden_dim / 4;
    dim3 block_dims(1024);
    dim3 grid_dims((total_count - 1) / 1024 + 1);  // (batch_size);

    fused_bias_residual<<<grid_dims, block_dims, 0, stream>>>(
        input, output, attn, bias, attn_bias, total_count, hidden_dim / 4, 1.0 / mp_size, preln);
}

template void launch_bias_residual<
    float>(float*, float*, float*, float*, float*, int, int, int, bool, hipStream_t);
template void launch_bias_residual<
    __half>(__half*, __half*, __half*, __half*, __half*, int, int, int, bool, hipStream_t);

__global__ void gptj_residual_add(float* input,
                                  float* output,
                                  float* attn,
                                  float* bias,
                                  float* attnbias,
                                  int total_count,
                                  int intermediate_size,
                                  float mp_scale)
{
    float4* input_cast = reinterpret_cast<float4*>(input);
    float4* output_cast = reinterpret_cast<float4*>(output);
    float4* attn_cast = reinterpret_cast<float4*>(attn);
    float4* bias_cast = reinterpret_cast<float4*>(bias);
    float4* attnbias_cast = reinterpret_cast<float4*>(attnbias);
    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    if (offset < total_count) {
        float4 data = input_cast[offset];
        float4 out = output_cast[offset];
        float4 res_vec = attn_cast[offset];
        float4 bias_data = bias_cast[offset % intermediate_size];

        if (attnbias) {
            float4 attn_bias = attnbias_cast[offset % intermediate_size];
            data.x += attn_bias.x;
            data.y += attn_bias.y;
            data.z += attn_bias.z;
            data.w += attn_bias.w;
        }
        data.x = out.x + res_vec.x + (data.x + bias_data.x) * mp_scale;
        data.y = out.y + res_vec.y + (data.y + bias_data.y) * mp_scale;
        data.z = out.z + res_vec.z + (data.z + bias_data.z) * mp_scale;
        data.w = out.w + res_vec.w + (data.w + bias_data.w) * mp_scale;

        input_cast[offset] = data;
    }
}

__global__ void gptj_residual_add(__half* input,
                                  __half* output,
                                  __half* attn,
                                  __half* bias,
                                  __half* attn_bias,
                                  int total_count,
                                  int intermediate_size,
                                  float mp_scale)
{
#ifdef HALF_PRECISION_AVAILABLE

    float2* input_cast = reinterpret_cast<float2*>(input);
    float2* output_cast = reinterpret_cast<float2*>(output);
    float2* attn_cast = reinterpret_cast<float2*>(attn);

    float2* bias_cast = reinterpret_cast<float2*>(bias);
    float2* attnbias_cast = reinterpret_cast<float2*>(attn_bias);

    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    if (offset < total_count) {
        float2 vals_vec = input_cast[offset];
        float2 out_vec = output_cast[offset];
        float2 res_vec = attn_cast[offset];

        float2 bias_vec = bias_cast[offset % intermediate_size];

        __half2* vals_half = reinterpret_cast<__half2*>(&vals_vec);
        __half2* out_half = reinterpret_cast<__half2*>(&out_vec);
        __half2* res_half = reinterpret_cast<__half2*>(&res_vec);
        __half2* bias_half = reinterpret_cast<__half2*>(&bias_vec);

        float2 low_data = __half22float2(vals_half[0]);
        float2 high_data = __half22float2(vals_half[1]);

        float2 low_out = __half22float2(out_half[0]);
        float2 high_out = __half22float2(out_half[1]);

        float2 low_res = __half22float2(res_half[0]);
        float2 high_res = __half22float2(res_half[1]);

        float2 low_bias = __half22float2(bias_half[0]);
        float2 high_bias = __half22float2(bias_half[1]);
        if (attn_bias) {
            float2 attn_bias_vec = attnbias_cast[offset % intermediate_size];
            __half2* attnbias_half = reinterpret_cast<__half2*>(&attn_bias_vec);
            float2 attn_low_bias = __half22float2(attnbias_half[0]);
            float2 attn_high_bias = __half22float2(attnbias_half[1]);
            low_data.x += attn_low_bias.x;
            low_data.y += attn_low_bias.y;
            high_data.x += attn_high_bias.x;
            high_data.y += attn_high_bias.y;
        }

        low_data.x = low_res.x + low_out.x + (low_data.x + low_bias.x) * mp_scale;
        low_data.y = low_res.y + low_out.y + (low_data.y + low_bias.y) * mp_scale;
        high_data.x = high_res.x + high_out.x + (high_data.x + high_bias.x) * mp_scale;
        high_data.y = high_res.y + high_out.y + (high_data.y + high_bias.y) * mp_scale;

        vals_half[0] = __float22half2_rn(low_data);
        vals_half[1] = __float22half2_rn(high_data);

        input_cast[offset] = vals_vec;
    }
#endif
}

template <typename T>
void launch_gptj_residual_add(T* input,
                              T* output,
                              T* attn,
                              T* bias,
                              T* attn_bias,
                              int hidden_dim,
                              int batch,
                              int mp_size,
                              hipStream_t stream)
{
    int total_count = batch * hidden_dim / 4;
    dim3 block_dims(1024);
    dim3 grid_dims((total_count - 1) / 1024 + 1);  // (batch_size);

    gptj_residual_add<<<grid_dims, block_dims, 0, stream>>>(
        input, output, attn, bias, attn_bias, total_count, hidden_dim / 4, 1.0 / mp_size);
}

template void launch_gptj_residual_add<float>(float*,
                                              float*,
                                              float*,
                                              float*,
                                              float*,
                                              int,
                                              int,
                                              int,
                                              hipStream_t);
template void launch_gptj_residual_add<__half>(__half*,
                                               __half*,
                                               __half*,
                                               __half*,
                                               __half*,
                                               int,
                                               int,
                                               int,
                                               hipStream_t);
template <typename T>
__global__ void moe_res_matmul(T* residual, T* coef, T* mlp_out, int seq_len, int hidden_dim)
{
    constexpr int granularity = 16;
    constexpr int vals_per_access = granularity / sizeof(T);

    T* residual_seq = residual + blockIdx.x * hidden_dim;
    T* mlp_out_seq = mlp_out + blockIdx.x * hidden_dim;

    for (unsigned tid = threadIdx.x * vals_per_access; tid < hidden_dim;
         tid += blockDim.x * vals_per_access) {
        T mlp[vals_per_access];
        T res[vals_per_access];
        T coef1[vals_per_access];
        T coef2[vals_per_access];

        mem_access::load_global<granularity>(mlp, mlp_out_seq + tid);
        mem_access::load_global<granularity>(res, residual_seq + tid);
        mem_access::load_global<granularity>(coef1, coef + tid);
        mem_access::load_global<granularity>(coef2, coef + tid + hidden_dim);

#pragma unroll
        for (int idx = 0; idx < vals_per_access; idx++) {
            mlp[idx] = mlp[idx] * coef2[idx] + res[idx] * coef1[idx];
        }

        mem_access::store_global<granularity>(mlp_out_seq + tid, mlp);
    }
}

template <typename T>
void launch_moe_res_matmul(T* residual,
                           T* coef,
                           T* mlp_out,
                           int seq_len,
                           int hidden_dim,
                           hipStream_t stream)
{
    dim3 grid_dim(seq_len);
    dim3 block_dim(1024);
    moe_res_matmul<<<grid_dim, block_dim, 0, stream>>>(
        residual, coef, mlp_out, seq_len, hidden_dim);
}

template void launch_moe_res_matmul(float* residual,
                                    float* coef,
                                    float* mlp_out,
                                    int seq_len,
                                    int hidden_dim,
                                    hipStream_t stream);
template void launch_moe_res_matmul(__half* residual,
                                    __half* coef,
                                    __half* mlp_out,
                                    int seq_len,
                                    int hidden_dim,
                                    hipStream_t stream);
